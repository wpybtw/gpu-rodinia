#include "hip/hip_runtime.h"
/*********************************************************************************
Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
All rights reserved.
  
Permission to use, copy, modify and distribute this software and its documentation for 
educational purpose is hereby granted without fee, provided that the above copyright 
notice and this permission notice appear in all copies of this software and that you do 
not sell the software.
  
THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
OTHERWISE.

The CUDA Kernel for Applying BFS on a loaded Graph. Created By Pawan Harish
**********************************************************************************/
#ifndef _KERNEL_H_
#define _KERNEL_H_

#define REG 4

__global__ void
Kernel(Node *g_graph_nodes, int *g_graph_edges, bool *g_graph_mask, bool *g_updating_graph_mask, bool *g_graph_visited, int *g_cost, int no_of_nodes)
{
	int tid = blockIdx.x * MAX_THREADS_PER_BLOCK + threadIdx.x;
	int r1, r2, r3, r4, r5, r6, r7, r8, r9, r10;
	if (tid < no_of_nodes && g_graph_mask[tid])
	{
		g_graph_mask[tid] = false;

		int k = g_graph_nodes[tid].starting;

		for (int i = 0; i < g_graph_nodes[tid].no_of_edges / REG; i++)
		{
			r1 = g_graph_edges[k++];
			r2 = g_graph_edges[k++];
			r3 = g_graph_edges[k++];
			r4 = g_graph_edges[k++];
			// r5 = g_graph_edges[k++];
			// r6 = g_graph_edges[k++];
			// r7 = g_graph_edges[k++];
			// r8 = g_graph_edges[k++];
			// r9 = g_graph_edges[k++];
			// r10 = g_graph_edges[k++];
			if (!g_graph_visited[r1])
			{
				g_cost[r1] = g_cost[tid] + 1;
				g_updating_graph_mask[r1] = true;
			}

			if (!g_graph_visited[r2])
			{
				g_cost[r2] = g_cost[tid] + 1;
				g_updating_graph_mask[r2] = true;
			}

			if (!g_graph_visited[r3])
			{
				g_cost[r3] = g_cost[tid] + 1;
				g_updating_graph_mask[r3] = true;
			}

			if (!g_graph_visited[r4])
			{
				g_cost[r4] = g_cost[tid] + 1;
				g_updating_graph_mask[r4] = true;
			}

			// if (!g_graph_visited[r5])
			// {
			// 	g_cost[r5] = g_cost[tid] + 1;
			// 	g_updating_graph_mask[r5] = true;
			// }

			// if (!g_graph_visited[r6])
			// {
			// 	g_cost[r6] = g_cost[tid] + 1;
			// 	g_updating_graph_mask[r6] = true;
			// }

			// if (!g_graph_visited[r7])
			// {
			// 	g_cost[r7] = g_cost[tid] + 1;
			// 	g_updating_graph_mask[r7] = true;
			// }

			// if (!g_graph_visited[r8])
			// {
			// 	g_cost[r8] = g_cost[tid] + 1;
			// 	g_updating_graph_mask[r8] = true;
			// }

			// if (!g_graph_visited[r9])
			// {
			// 	g_cost[r9] = g_cost[tid] + 1;
			// 	g_updating_graph_mask[r9] = true;
			// }

			// if (!g_graph_visited[r10])
			// {
			// 	g_cost[r10] = g_cost[tid] + 1;
			// 	g_updating_graph_mask[r10] = true;
			// }
		}

		for (size_t i = k; i < (g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting); i++)
		{
			int id = g_graph_edges[i];
			if (!g_graph_visited[id])
			{
				g_cost[id] = g_cost[tid] + 1;
				g_updating_graph_mask[id] = true;
			}
		}
	}
}

#endif
